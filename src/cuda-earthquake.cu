#include "hip/hip_runtime.h"
/*
 * Diego Mazzieri 0000792583
 * 
 * Versione CUDA del programma earthquake.c
 */

#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>     /* rand() */
#include <assert.h>

/* energia massima */
#define EMAX 4.0f
/* energia da aggiungere ad ogni timestep */
#define EDELTA 1e-4
/* dimensione nel caso di thread block 2D */
#define BLKDIM 32
/* dimensione nel caso di thread block 1D */
#define BLKDIM_LINEAR (BLKDIM*BLKDIM)

/**
 * Restituisce un puntatore all'elemento di coordinate (i,j) del
 * dominio grid con n colonne.
 * Utile sia all'host (per fare setup) che al device (per fare increment_energy e propagate_energy).
 */
__device__ __host__ static inline float *IDX( float *grid, int i, int j, int n )
{
    return (grid + i*n + j);
}

/**
 * Restituisce un numero reale pseudocasuale con probabilita' uniforme
 * nell'intervallo [a, b], con a < b.
 */
__host__ float randab( float a, float b )
{
    return a + (b-a)*(rand() / (float)RAND_MAX);
}

/**
 * Inizializza il dominio grid di dimensioni n*n con, nelle celle intermedie, valori di energia
 * scelti con probabilità uniforme nell'intervallo [fmin, fmax], nelle celle ai bordi valori nulli.
 * 
 * |0 0 0 0 0 0 0 0 0 0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0|# # # # # # # #|0|
 * |0 0 0 0 0 0 0 0 0 0|
 * 
 * Non è stato possibile parallelizzare questa funzione con CUDA in quanto rand() non e' thread-safe.
 */
__host__ void setup( float* grid, int ext_n, float fmin, float fmax )
{
    /* La prima riga della matrice è costituita da sole ghost cells */
    for( int j=0; j<ext_n; j++) {
        *IDX(grid, 0, j, ext_n) = 0;
    }
    /* Le successive n-2 righe iniziano e finiscono con una ghost cell 
       mentre hanno valori casuali nelle celle intermedie. */
    for ( int i=1; i<ext_n-1; i++ ) {
        *IDX(grid, i, 0, ext_n) = 0;
        for ( int j=1; j<ext_n-1; j++ ) {
            *IDX(grid, i, j, ext_n) = randab(fmin, fmax);
        }
        *IDX(grid, i, ext_n-1, ext_n) = 0;
    }
    /* Anche l'ultima riga, come la prima, è costituita da sole ghost cells */
    for( int j=0; j<ext_n; j++) {
        *IDX(grid, ext_n-1, j, ext_n) = 0;
    }
}

/**
 * Somma delta a tutte le celle tranne quelle ai bordi del dominio grid di dimensioni
 * n*n. Questo kernel realizza il passo 1 descritto nella specifica del progetto.
 */
 __global__ void increment_energy( float *grid, int ext_n, float delta )
{
    /* Partendo da ext_n salto automaticamente la prima riga composta da sole ghost cells */
    const int i = ext_n + threadIdx.x + blockIdx.x * blockDim.x;
    const int col = i % ext_n;
    
    /* Escludo le ghost cells o le celle al di fuori del dominio dall'incremento */
    if (col > 0 && /* non appartenente alla prima colonna */ 
        col < ext_n-1 && /* non appartenente all'ultima colonna */
        i < ext_n*(ext_n-1)) { /* prima dell'ultima riga */
        grid[i] += delta;
    }
}

/**
 * Calcola il numero di celle la cui energia è strettamente
 * maggiore di EMAX e inserisce il risultato in count.
 */
__global__ void count_cells( float *grid, int ext_size, int *count )
{
    __shared__ int local_sum[BLKDIM_LINEAR];
    int lindex = threadIdx.x;
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int bsize = blockDim.x / 2;
    /* 1 -> la cella ha energia maggiore della massima 
       0 -> la cella ha energia minore della massima o l'accesso è out of bound */
    local_sum[lindex] = (gindex < ext_size && grid[gindex] > EMAX);
    __syncthreads();
    /* Riduzione dell'array local_sum */
    while ( bsize > 0 ) {
        if ( lindex < bsize ) {
            local_sum[lindex] += local_sum[lindex + bsize];
        }
        bsize /= 2;
        __syncthreads();
    }
    if ( lindex == 0 ) {
        /* Aggiungo atomicamente a count il risultato della riduzione calcolata nel blocco corrente */
        atomicAdd(count, local_sum[0]);
    }
}

/**
 * Calcola l'energia totale della grid sommando 
 * il valore delle singole celle e inserisce il risultato in sum. 
 */
__global__ void sum_energy( float *grid, int ext_size, float *sum )
{
    __shared__ float local_sum[BLKDIM_LINEAR];
    int lindex = threadIdx.x;
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int bsize = blockDim.x / 2;
    /* 0 -> accesso out of bound */
    local_sum[lindex] = (gindex < ext_size) ? grid[gindex] : 0;
    __syncthreads();
    while ( bsize > 0 ) {
        if ( lindex < bsize ) {
            local_sum[lindex] += local_sum[lindex + bsize];
        }
        bsize /= 2;
        __syncthreads();
    }
    if ( lindex == 0 ) {
        atomicAdd(sum, local_sum[0]);
    }
}

/**
 * Sostituisce le energie cinetiche totali calcolate per ogni step
 * con le rispettive energie cinetiche medie.
 */
__global__ void average_energy( float *sums, int nsteps, int size ) 
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < nsteps ) {
        sums[i] /= size;
    }
}

/** 
 * Distribuisce l'energia di ogni cella a quelle adiacenti.
 * cur denota il dominio corrente, next denota il dominio
 * che conterra' il nuovo valore delle energie. Questa funzione
 * realizza il passo 2 descritto nella specifica del progetto.
 */
__global__ void propagate_energy( float *cur, float *next, int ext_n )
{
    /* Prevedendo una ghost area anche per la memoria shared 
       vengono aggiornate (BLKDIM-2)*(BLKDIM-2) celle 
       ma è possibile semplificare la gestione delle celle ai bordi 
       del dominio gestito da ogni blocco. */
    __shared__ float buf[BLKDIM][BLKDIM];
    
    /* Gli indici globali vanno quindi calcolati come se i blocchi
       fossero di dimensione (BLKDIM-2)*(BLKDIM-2) */  
    const int gi = threadIdx.y + blockIdx.y * (blockDim.y-2);
    const int gj = threadIdx.x + blockIdx.x * (blockDim.x-2);

    const int li = threadIdx.y;
    const int lj = threadIdx.x;
   
    if ( gi < ext_n && gj < ext_n ) {
        buf[li][lj] = *IDX(cur, gi, gj, ext_n);
        float *out = IDX(next, gi, gj, ext_n);
        __syncthreads();
        const float FDELTA = EMAX/4;
        /* Escludere le ghost cells del buf con il controllo su li e lj
           basterebbe nel caso in cui n fosse multiplo di BLKSIZE-2;
           ma per funzionare nel caso generale, devo anche controllare
           che non vengano considerate celle al di fuori del dominio. */
        if ((li > 0) && (li < blockDim.y-1) &&
            (lj > 0) && (lj < blockDim.x-1) &&
            (gi < ext_n-1) && (gj < ext_n-1)) {
            float F = buf[li][lj];
            if (buf[li  ][lj-1] > EMAX) { F += FDELTA; }
            if (buf[li  ][lj+1] > EMAX) { F += FDELTA; }
            if (buf[li-1][lj  ] > EMAX) { F += FDELTA; }
            if (buf[li+1][lj  ] > EMAX) { F += FDELTA; }
    
            if (F > EMAX) {
                F -= EMAX;
            }
    
            *out = F;
        }
    }
}

int main( int argc, char* argv[] )
{
    float *grid;
    float *d_cur, *d_next, *d_sum, *Emean;
    int s, n = 256, ext_n, nsteps = 2048;
    int *c, *d_c;
    srand(19);

    if ( argc > 3 ) {
        fprintf(stderr, "Usage: %s [nsteps [n]]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        nsteps = atoi(argv[1]);
    }

    if ( argc > 2 ) {
        n = atoi(argv[2]);
    }

    /* Parametri in input non validi */
    if ( nsteps <= 0 || n <= 0 ) {
        /* Se non vengono effettuate computazioni 
           il tempo di esecuzione è considerato nullo */
    #ifdef BENCHMARK
	    printf("%.4f\n", 0.0);
    #endif
        return EXIT_FAILURE;
    }

    /* Includo le ghost cells nel dominio */
    ext_n = n+2;
    const size_t grid_mem_size = ext_n*ext_n*sizeof(float);
    const size_t c_mem_size = nsteps*sizeof(int);
    const size_t sum_mem_size = nsteps*sizeof(float);
    
    /* Per evitare comunicazioni ad ogni timestep tra GPU e CPU, 
       memorizzo il numero di celle maggiori di EMAX e l'energia media 
       all'interno di due array allocati nella GPU, che trasferisco nella CPU
       solo una volta che la computazione dello stencil è completata */
    hipMalloc((void **)&d_c, c_mem_size);
    hipMalloc((void **)&d_sum, sum_mem_size);
    hipMalloc((void **)&d_cur, grid_mem_size);
    hipMalloc((void **)&d_next, grid_mem_size);

    c = (int*)malloc(c_mem_size); assert(c);
    Emean = (float*)malloc(sum_mem_size); assert(Emean);
    grid = (float*)malloc(grid_mem_size); assert(grid);

    /* Inizializzo a 0 gli array allocati nella GPU */
    hipMemset(d_c, 0, c_mem_size);
    hipMemset(d_sum, 0, sum_mem_size);

    setup(grid, ext_n, 0, EMAX*0.1);

    /* Copio la grid inizializzata dalla CPU alla GPU */
    hipMemcpy(d_cur, grid, grid_mem_size, hipMemcpyHostToDevice);

    dim3 linearBlock(BLKDIM_LINEAR);
    dim3 squaredBlock(BLKDIM, BLKDIM);
    dim3 linearGrid((ext_n*ext_n + BLKDIM_LINEAR - 1) / BLKDIM_LINEAR);
    dim3 squaredGridGhost((ext_n + BLKDIM - 3) / (BLKDIM - 2), (ext_n + BLKDIM - 3) / (BLKDIM - 2));
    
    const double tstart = hpc_gettime();
    for (s=0; s<nsteps; s++) {
        /* Per utilizzare il minor numero possibile di blocchi considero un partizionamento 1D */
        increment_energy<<<linearGrid, linearBlock>>>(d_cur, ext_n, EDELTA);
        /* Dovendo effettuare l'operazione di riduzione su una matrice,
           considero un partizionamento 1D del dominio trattandola come fosse un array */ 
        count_cells<<<linearGrid, linearBlock>>>(d_cur, ext_n*ext_n, d_c + s);
        /* Il numero di blocchi va calcolato tenendo in considerazione che 
           la shared memory fa uso anch'essa di ghost cells, quindi ogni blocco 
           aggiorna (BLKDIM-2)*(BLKDIM-2) celle pur avendo thread block di dimensione BLKDIM*BLKDIM*/
        propagate_energy<<<squaredGridGhost, squaredBlock>>>(d_cur, d_next, ext_n);
        /* Stesso ragionamento utilizzato per count_cells */
        sum_energy<<<linearGrid, linearBlock>>>(d_next, ext_n*ext_n, d_sum + s);

        float *tmp = d_cur;
        d_cur = d_next;
        d_next = tmp;
    }
    /* Prima di trasferire alla CPU i risultati sostituisco 
       all'energia cinetica totale calcolata ad ogni iterazione
       la rispettiva energia cinetica media */
    average_energy<<<(nsteps + BLKDIM_LINEAR - 1) / BLKDIM_LINEAR, linearBlock>>>(d_sum, nsteps, n*n);
    hipMemcpy(c, d_c, c_mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(Emean, d_sum, sum_mem_size, hipMemcpyDeviceToHost);
    /* Non è stato necessario utilizzare hipDeviceSynchronize in quanto hipMemcpy, implicitamente, 
       blocca la CPU fino a quando le precedenti chiamate CUDA non sono state completate */
    const double elapsed = hpc_gettime() - tstart;

#ifndef BENCHMARK
    for (s=0; s<nsteps; s++) {
        printf("%d %f\n", c[s], Emean[s]);
    }
    double Mupdates = (((double)n)*n/1.0e6)*nsteps; /* milioni di celle aggiornate per ogni secondo di wall clock time */
    fprintf(stderr, "%s : %.4f Mupdates in %.4f seconds (%f Mupd/sec)\n", argv[0], Mupdates, elapsed, Mupdates/elapsed);
#else
    printf("%.4f\n", elapsed);
#endif

    free(grid);
    free(c);
    free(Emean);
    hipFree(d_cur);
    hipFree(d_next);
    hipFree(d_c);
    hipFree(d_sum);

    return EXIT_SUCCESS;
}
